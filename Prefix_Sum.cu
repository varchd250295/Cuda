
#include <hip/hip_runtime.h>
/*
#define __CUDACC__
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdio.h>
#include <conio.h>
#define Block_Size 4

__global__ void PrefixSumEfficient(int *A_d)
{
	int stride,index,i;
	__shared__ int XY[Block_Size*2];

	i=threadIdx.x;
	if(i<2*Block_Size)	//For prefix sum we take block size as half the input size
		XY[i]=A_d[i];	//Load from global memory to shared memory
	__syncthreads();

	for(stride=1; stride<=Block_Size; stride*=2)	//Reduction Phase
	{
		index=((threadIdx.x + 1)*stride*2)-1;
		if(index<2*Block_Size)
			XY[index]+=XY[index-stride];
		__syncthreads();
	}

	for(stride=Block_Size/2; stride>0; stride/=2)	//Post Reduction Phase
	{
		__syncthreads();
		index=((threadIdx.x + 1)*stride*2)-1;
		if(index+stride<2*Block_Size)
			XY[index+stride]+=XY[index];
	}

	__syncthreads();
	A_d[i]=XY[i];
}

int main()
{
	int i, *A_d, *A, tile;

	int size=sizeof(int)*Block_Size*2;

	A=(int *)malloc(size);

	printf("Elements to be added:\n");
	for(i=0; i<Block_Size*2; i++)
		printf("%d \t",A[i]=i+1);

	cudaMalloc((void**)&A_d,size);
	cudaMemcpy(A_d,A,size,cudaMemcpyHostToDevice);
		PrefixSumEfficient<<<1,Block_Size*2>>>(A_d);
	cudaMemcpy(A,A_d,size,cudaMemcpyDeviceToHost);
	printf("\nEfficient:\n");
	for(i=0; i<Block_Size*2; i++)
		printf("%d	",A[i]);

	cudaFree(A_d);

	getch();
	return 0;
}
*/

/* OUTPUT -

Elements to be added:
1       2       3       4       5       6       7       8
Efficient:
1       3       6       10      15      21      28      36

*/
