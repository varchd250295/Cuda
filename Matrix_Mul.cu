
#include <hip/hip_runtime.h>
/*

#ifndef __CUDACC__
#define __CUDACC__
#endif


#include "cuda_runtime.h"
#include "device_launch_parameters.h"


#include <stdio.h>
#include <conio.h>

const int TILE_WIDTH=2;
const int width=4;

__global__ void matrixmul(int *d_M,int *d_N,int *d_P)
{
	__shared__ int dS_M[TILE_WIDTH][TILE_WIDTH];
	__shared__ int dS_N[TILE_WIDTH][TILE_WIDTH];
	
	int by= blockIdx.y;
	int ty=threadIdx.y;
	int bx=blockIdx.x;
	int tx=threadIdx.x;

	int Row= by*TILE_WIDTH+ty;
	int Col= bx*TILE_WIDTH+tx;
	int pvalue=0;
	for(int m=0;m<(width/TILE_WIDTH);m++)
	{   dS_M[ty][tx]=d_M[Row*width + (m*TILE_WIDTH+tx)] ;
	    dS_N[ty][tx]=d_N[Col+(m*TILE_WIDTH+ty)*width] ;
		__syncthreads();
		for(int k=0;k<TILE_WIDTH;k++)
		{
			pvalue += dS_M[ty][k]*dS_N[k][tx] ;
			
		__syncthreads();
		}
		d_P [Row*width +Col]=pvalue;

	}

}
int main()
{
	int i,j,size;
	int h_M[width][width],h_N[width][width],h_P[width][width];
	int *d_M,*d_N,*d_P;
	
	printf("\n Enter according to width= %d \n ", width);
	
	for(i=0;i<width;i++)
	{
		for(j=0;j<width;j++)
			{ h_M[i][j]=2;
		      h_N[i][j]=2;
		     if(i==j)
		       {
				   h_N[i][j]= 1;
			     h_M[i][j]= 1;
			  }
   }
	}

	

	printf("\nh_M array : \n");
	for(i=0;i<width;i++)
	{   printf("\n");
		for(j=0;j<width;j++)
			printf("%d ",h_M[i][j]);
	}
	
	printf("\n\nh_N array : \n");
	for(i=0;i<width;i++)
	{   printf("\n");
		for(j=0;j<width;j++)
			printf("%d ",h_N[i][j]);
	}
	
	

	size=sizeof(int)*width*width;
	cudaMalloc((void**)&d_M,size);
	cudaMalloc((void**)&d_N,size);
	cudaMalloc((void**)&d_P,size);
	cudaMemcpy(d_M,h_M,size,cudaMemcpyHostToDevice);
	cudaMemcpy(d_N,h_N,size,cudaMemcpyHostToDevice);
	dim3 dimGrid((width/TILE_WIDTH),(width/TILE_WIDTH),1);
	dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);

	matrixmul<<<dimGrid,dimBlock>>>(d_M,d_N,d_P);
	
	cudaMemcpy(h_P,d_P,size,cudaMemcpyDeviceToHost);
	cudaFree(d_M);
	cudaFree(d_N);
	cudaFree(d_P);

	printf("\n Resultant Array ( h_M * h_N ): \n");
	for(i=0;i<width;i++)
	{   printf("\n");
		for(j=0;j<width;j++)
			printf("%d ",h_P[i][j]);
	}
	
  getch();
  return 0;
}

*/