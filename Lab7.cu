
#include <hip/hip_runtime.h>
/*
#ifndef __CUDACC__
#define __CUDACC__
#endif

#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>

__constant__ int M[3];

__global__ void convolution_1D_basic(int *N, int *P,int Mask_Width, int Width) 
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int Pvalue = 0;
	int N_start_point = i - (Mask_Width/2);
	for (int j = 0; j < Mask_Width; j++) 
	{
		if (N_start_point + j >=0 && N_start_point + j < Width) 
		{
			Pvalue += N[N_start_point + j]*M[j];
		}
	}
	P[i] = Pvalue;
} 

int main()
{
	const int width=5;
	int i,j,size;
	int h_M[width],h_N[width],h_P[width];
	int *d_M,*d_N,*d_P;
	
	printf("\n width= %d \n ", width);

	
	
	h_N[0]=0;
	h_N[1]=1;
	h_N[2]=0;

	printf("\n Enter elements \n", width);

	for(i=0;i<width;i++)
		scanf("%d\n",h_M[i]);

	printf("\n");
	
	size=sizeof(int)*width;
	cudaMalloc((void**)&d_M,size);
	cudaMalloc((void**)&d_N,size);
	cudaMalloc((void**)&d_P,size);

	cudaMemcpyToSymbol(M,h_N,size);

	

	convolution_1D_basic<<<1,width>>>(d_N,d_P,size,width);
	
	cudaMemcpy(h_P,d_P,size,cudaMemcpyDeviceToHost);

	cudaFree(d_N);

	cudaFree(d_P);

	printf("\n\nRESULT : \n");

	for(j=0;j<width;j++)
		printf("%d ",h_P[j]);
	
	getch();
	return 0;

}
*/