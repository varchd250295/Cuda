
#include <hip/hip_runtime.h>
/*
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <conio.h>
#include <stdio.h>

__constant__ int d_m[6];

__global__ void convolution(int *N,int *P,int maskwidth,int width)
{
    int i =blockIdx.x*blockDim.x+ threadIdx.x;
	int pvalue=0;
	int startpt=i-(maskwidth/2);

	for(int j=0;j<maskwidth;j++)
	{
          if(startpt+j >=0 && startpt+j<width)
		{
              pvalue+=N[startpt+j]*d_m[j];
		}
	}
	P[i]=pvalue;
}
 
int main()   
{
       int i,n1=6,mw=3,n[6]={1,2,3,4,5,6},m[3]={1,2,3},p[6],*n_d,*p_d;
       int size=sizeof(int)*n1;

	   printf(" ORIGINAL ARRAY \n");
	   for(i=0;i<6;i++)
	       printf("%d \t",n[i]);

	   printf("\n MASK ARRAY \n");
	   for(i=0;i<3;i++)
	       printf("%d \t",m[i]);

	cudaMalloc((void**)&n_d,size);
       cudaMalloc((void**)&p_d,size);
       cudaMemcpy(n_d,n,size,cudaMemcpyHostToDevice);
               cudaMemcpyToSymbol(d_m,m,size);

		//cudaMemcpy(p_d,p,size,cudaMemcpyHostToDevice);

		convolution<<<1,n1>>>(n_d,p_d,mw,n1); 
        cudaMemcpy(p,p_d,size,cudaMemcpyDeviceToHost);

		printf("\n Array after convolution is:\n");

        for(i=0;i<n1;i++)
	       printf("%d \t",p[i]);

        cudaFree(n_d);
		cudaFree(d_m);
		cudaFree(p_d);
		getch();
		return 0;
}
*/

/* OUTPUT -

 ORIGINAL ARRAY
1       2       3       4       5       6
 MASK ARRAY
1       2       3
 Array after convolution is:
8       14      20      26      32      17


*/
