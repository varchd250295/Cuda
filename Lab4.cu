
#include <hip/hip_runtime.h>
/*
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdio.h>

int main( void ) {
cudaDeviceProp  prop;
int count;
cudaGetDeviceCount( &count ) ;
    for (int i=0; i< count; i++) {
cudaGetDeviceProperties( &prop, i ) ;
printf( "   --- General Information for device %d ---\n", i );
printf( "Name:  %s\n", prop.name );
printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
printf( "Clock rate:  %d\n", prop.clockRate );
printf( "Device copy overlap:  " );
        if (prop.deviceOverlap)
printf( "Enabled\n" );
        else
printf( "Disabled\n");
printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
printf( "Enabled\n" );
        else
printf( "Disabled\n" );

printf( "   --- Memory Information for device %d ---\n", i );
printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
printf( "Max mem pitch:  %ld\n", prop.memPitch );
printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

printf( "   --- MP Information for device %d ---\n", i );
printf( "Multiprocessor count:  %d\n",  prop.multiProcessorCount );
printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
printf( "Registers per mp:  %d\n", prop.regsPerBlock );
printf( "Threads in warp:  %d\n", prop.warpSize );
printf( "Max threads per block:  %d\n", prop.maxThreadsPerBlock );
printf( "Max thread dimensions:  (%d, %d, %d)\n",   prop.maxThreadsDim[0], prop.maxThreadsDim[1],  prop.maxThreadsDim[2] );
printf( "Max grid dimensions:  (%d, %d, %d)\n",    prop.maxGridSize[0], prop.maxGridSize[1],     prop.maxGridSize[2] );
printf( "\n" );
    }
}
*/



/*
--- General Information for device 0 ---
Name:  GeForce GTX 950M
Compute capability:  5.0
Clock rate:  1124000
Device copy overlap:  Enabled
Kernel execution timeout :  Enabled
   --- Memory Information for device 0 ---
Total global mem:  -2147483648
Total constant Mem:  65536
Max mem pitch:  2147483647
Texture Alignment:  512
   --- MP Information for device 0 ---
Multiprocessor count:  5
Shared mem per mp:  49152
Registers per mp:  65536
Threads in warp:  32
Max threads per block:  1024
Max thread dimensions:  (1024, 1024, 64)
Max grid dimensions:  (2147483647, 65535, 65535)

Press any key to continue . . .
*/