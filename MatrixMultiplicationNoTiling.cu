
#include "hip/hip_runtime.h"

#include <stdlib.h>

#include <stdio.h>
#define blockSize 16

__global__ void multiply(int *d_a,int *d_b,int *d_c,int n)
{
	int i;
	int temp=0;
    
	int row=blockIdx.y*blockDim.y + threadIdx.y;
	int column=blockIdx.x*blockDim.x + threadIdx.x;

	for(i=0;i<n;i++)
	temp+=d_a[row*n+i]*d_b[i*n+column];

	d_c[row*n+column]=temp;
}


int main()
{
	int n,i,j;
	int *h_a,*h_b,*h_c;
	int *d_a,*d_b,*d_c;
	int size;
	int temp=0;


	printf("Enter the size of the matrix :\n");
	scanf("%d",&n);

	h_a=(int *)malloc((size=(sizeof(int)*n*n)));
	h_b=(int *)malloc(size);
	h_c=(int *)malloc(size);

	dim3 grid(blockSize,blockSize);
	dim3 block((n-1)/blockSize +1,(n-1)/blockSize +1);

    hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);


	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			h_a[i*n+j]=++temp;
			h_b[i*n+j]=1;
		}
	}


	hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,size,hipMemcpyHostToDevice);

	multiply<<<grid,block>>>(d_a,d_b,d_c,n);

	hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);

	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		printf("%d \n",h_c[i*n+j]);

	}

	free(h_a);
	free(h_b);
	free(h_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	

    return 0;
}
