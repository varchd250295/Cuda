
#include <hip/hip_runtime.h>
/*
#ifndef __CUDACC__
#define __CUDACC__
#endif

#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>

static const int wholeArraySize = 100000000;
static const int blockSize = 16;
static const int gridSize = 4; //this number is hardware-dependent; usually #SM*2 is a good number.

__global__ void sumCommMultiBlock(const int *gArr, int arraySize, int *gOut) 
{
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    int sum = 0;
    for (int i = gthIdx; i < arraySize; i += gridSize)
        sum += gArr[i];
    __shared__ int shArr[blockSize];
    shArr[thIdx] = sum;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size)
            shArr[thIdx] += shArr[thIdx+size];
        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = shArr[0];
}

__host__ int sumArray(int* arr) 
{
    int* dev_arr;
    cudaMalloc((void**)&dev_arr, wholeArraySize * sizeof(int));
    cudaMemcpy(dev_arr, arr, wholeArraySize * sizeof(int), cudaMemcpyHostToDevice);

    int out;
    int* dev_out;
    cudaMalloc((void**)&dev_out, sizeof(int)*gridSize);
    
    sumCommMultiBlock<<<gridSize, blockSize>>>(dev_arr, wholeArraySize, dev_out);
    //dev_out now holds the partial result
    sumCommMultiBlock<<<1, blockSize>>>(dev_out, gridSize, dev_out);
    //dev_out[0] now holds the final result
    cudaDeviceSynchronize();
    
    cudaMemcpy(&out, dev_out, sizeof(int), cudaMemcpyDeviceToHost);
    cudaFree(dev_arr);
    cudaFree(dev_out);
    return out;
}

// REDUCTION SUM = 2 METHODS

// CONVOLUTION 

*/