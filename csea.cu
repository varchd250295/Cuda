#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/device_functions.h>
#include <string.h>
#define SIZE 100



__global__ void rows(int a[],int b[],int r,int c,int row)
{
	int i=threadIdx.x;
	int suma=0,sumb=0;

	for(int j=0;j<c;j++)
	{
		suma+=a[i*r+j];
	}

	for(int j=0;j<r;j++)
	{
		sumb=0;

		for(int k=0;k<c;k++)
		{
			sumb+=b[j*r+k];
		}

		if(suma==sumb)
			row++;

	}

}

__global__ void ele(int a[],int b[],int r,int c)
{





}



int main()
{
	int a[SIZE],b[SIZE],r=0,c=0,i=0,j=0;
	int *d_a,*d_b,*d_r;
	int row[SIZE];

	printf("Enter the no. of rows and columns : \n");
	scanf("%d\n%d",&r,&c);

	printf("Enter the 1st array elements : \n");
	for(i=0;i<r;i++)
		for(j=0;j<c;j++)
			scanf("%d",&a[r*i+j]);

	printf("Enter the 2nd array elements : \n");
	for(i=0;i<r;i++)
		for(j=0;j<c;j++)
			scanf("%d",&b[r*i+j]);


	
	
	int size=sizeof(int)*(r*c);

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void**)&d_r,sizeof(int)*r);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_r,row,sizeof(int),hipMemcpyHostToDevice);

	rows<<<1,r>>>(d_a,a,r,c,row);

	printf("No. of same rows = %d",row);

	

	hipFree(d_a);
	hipFree(d_b);


}