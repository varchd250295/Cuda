
#include <hip/hip_runtime.h>
/*
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include<stdio.h>
#include<stdlib.h>
#include<conio.h>

__global__ void convolution(int *N, int *M, int *P, int mask_width, int width)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int pvalue=0;
	int N_start_point=i-(mask_width/2);
	
	for(int j=0; j<mask_width; j++)
	{
		if(N_start_point+j>=0 && N_start_point+j<width)
			pvalue +=N[N_start_point+ j]*M[j];
	}
	P[i]=pvalue;
}

int main()
{
	int i, n, m, *A, *B, *C, *N, *M, *P;
	
	n=8;	
	m=5;

	A=(int *)malloc(sizeof(int)*n);
	B=(int *)malloc(sizeof(int)*m);
	C=(int *)malloc(sizeof(int)*n);

	printf("\n\nElements in A:\n");

	for(i=0; i<n; i++)
		printf("%d \t",A[i]=i+1);

	printf("\n\nElements in B:\n");

	for(i=0; i<m; i++)
		printf("%d \t",B[i]=i+1);

	cudaMalloc((void**)&N,sizeof(int)*n);
	cudaMalloc((void**)&M,sizeof(int)*m);
	cudaMalloc((void**)&P,sizeof(int)*n);

	cudaMemcpy(N,A,sizeof(int)*n,cudaMemcpyHostToDevice);
	cudaMemcpy(M,B,sizeof(int)*m,cudaMemcpyHostToDevice);

	dim3 DimGrid(1,1,1);
	dim3 DimBlock(n,1,1);
	convolution<<<DimGrid,DimBlock>>>(N,M,P,m,n);

	cudaMemcpy(C,P,sizeof(int)*n,cudaMemcpyDeviceToHost);

	printf("\n\nOutput:\n");

	for(i=0; i<n; i++)
		printf("%d \t",C[i]);
	printf("\n");

	cudaFree(N);
	cudaFree(M);
	cudaFree(P);
	getch();
	return 0;
}
*/

/* OUTPUT - 


Elements in A:
1       2       3       4       5       6       7       8

Elements in B:
1       2       3       4       5

Output:
26      40      55      70      85      100     70      44


*/