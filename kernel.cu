
#include <hip/hip_runtime.h>
/*

#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>



__global__ void addMultipleBlocks(float *d_a,float *d_b,float *d_c,int m,int n)
{
	int i=blockIdx.x*blockDim.x+ threadIdx.x;

	if(i<(m*n))
		d_c[i]=d_a[i]+d_b[i];

}


__global__ void addSingleBlock(float *d_a,float *d_b,float *d_c,int m,int n)
{
	
	int i=threadIdx.x;

	d_c[i]=d_a[i]+d_b[i];

}



int main()
{
	float *a,*b,*c;

	float *d_a,*d_b,*d_c;
	int size,m,n,i,j;
	time_t start;

	printf("Enter the number of rows and columns : \n");

	scanf("%d%d",&m,&n);

	a=(float *)malloc((size=sizeof(float)*m*n));
	b=(float *)malloc(size);
	c=(float *)malloc(size);


	for(i=0;i<m;i++)
		for(j=0;j<n;j++)
		{
			a[i*n+j]=i*2+j+0.8;
			b[i*n+j]=i*2+j+0.1;
		}

	cudaMalloc((void **)&d_a,size);
	cudaMalloc((void**)&d_b,size);
	cudaMalloc((void **)&d_c,size);

	cudaMemcpy(d_a,a,size,cudaMemcpyHostToDevice);
	cudaMemcpy(d_b,b,size,cudaMemcpyHostToDevice);

	start=time(0);

	addSingleBlock<<<1,(m*n)>>>(d_a,d_b,d_c,m,n);

	cudaMemcpy(c,d_c,size,cudaMemcpyDeviceToHost);

	printf("Result using a single block : \n");


	for(i=0;i<m;i++)
	{
		for(j=0;j<n;j++)
		printf("%f ",c[i*n+j]);

		printf("\n");
	}
	printf("\n\n\n TIME TAKEN TO COMPUTER THE ADDITION WITH SINGLE BLOCK - %f",(difftime(time(0),start)));

	cudaFree(d_c);
	free(c);

	cudaMalloc((void **)&d_c,size);
	c=(float *)malloc(size);

	start=time(0);

	addMultipleBlocks<<< (int)((m*n)/9.0+1),9>>>(d_a,d_b,d_c,m,n);

	cudaMemcpy(c,d_c,size,cudaMemcpyDeviceToHost);

	printf("Result using a multiple blocks(3x3) : \n");


	for(i=0;i<m;i++)
	{
		for(j=0;j<n;j++)
		printf("%f ",c[i*n+j]);

		printf("\n");
	}

	printf("\n\n\n TIME TAKEN TO COMPUTER THE ADDITION WITH	MULTIPLE BLOCKS - %f",(difftime(time(0),start)));


	
	free(a);
	free(b);
	free(c);
	
	cudaFree(d_a);
	cudaFree(d_b);
	cudaFree(d_c);



	return 0;
}

*/