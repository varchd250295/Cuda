
#include <hip/hip_runtime.h>
/*

#ifndef __CUDACC__
#define __CUDACC__
#endif
#include "cuda_runtime.h"
#include "device_launch_parameters.h"

#include<stdio.h>
#include<conio.h>
#define width 4

__global__ void sumReductionEff(int *d_pSum, int n)
{
	int i = 0;
	__shared__ int partialSum[width];
	unsigned int t=threadIdx.x;
	partialSum[t] = d_pSum[t];
	__syncthreads();
	for(unsigned int stride=blockDim.x/2; stride>0; stride/=2)
	{
		__syncthreads();
		if(t<stride)
			partialSum[t] += partialSum[t+stride];
	}
	d_pSum[t] = partialSum[t];
}

__global__ void sumReductionIneff(int *d_pSum, int n)
{
	int i = 0;
	__shared__ int partialSum[width];
	unsigned int t=threadIdx.x;
	partialSum[t] = d_pSum[t];
	__syncthreads();
	for(unsigned int stride=1; stride<blockDim.x; stride*=2)
	{
		__syncthreads();
		if(t%2 == 0)
			partialSum[t] += partialSum[t+stride];
	}
	d_pSum[t] = partialSum[t];
}




int main()
{
	int pSum[] = {1,2,3,4,5,6,7,8};
	int n = 8, *d_pSum, i = 0;
	int size = n*sizeof(int);
	cudaMalloc((void**)&d_pSum, size);
	cudaMemcpy(d_pSum, pSum, size, cudaMemcpyHostToDevice);
	printf("\n Elements of the array : \n");
	for(i=0; i<n; i++)
		printf("%d \t", pSum[i]);
	sumReductionIneff<<<1, n>>>(d_pSum, n);
	cudaMemcpy(pSum, d_pSum, size, cudaMemcpyDeviceToHost);
	printf("\n");
	printf("Sum of the array elements = %d", pSum[0]);
	getch();

	return 0;
}

*/