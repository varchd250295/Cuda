
#include <hip/hip_runtime.h>
/*

#ifndef __CUDACC__
#define __CUDACC__
#endif

#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <stdio.h>
#include <stdlib.h>*
#include <conio.h>

const int TILE_WIDTH=2;


__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

	// Identify the row and column of the Pd element to work on
	int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
	int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
	float Pvalue = 0;

	// Loop over the Md and Nd tiles required to compute the Pd element
	for (int m = 0; m < Width/TILE_WIDTH; ++m) 
	{

		// Collaborative loading of Md and Nd tiles into shared memory
		Mds[threadIdx.y][threadIdx.x] = Md[Row*Width + (m*TILE_WIDTH + threadIdx.x)];
		Nds[threadIdx.y][threadIdx.x] = Nd[Col + (m*TILE_WIDTH + threadIdx.y)*Width];
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k)
		Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
		__syncthreads();
	}

	Pd[Row*Width+Col] = Pvalue;
}

int main()
{
	const int width=4;
	int i,j,size;
	float h_M[width][width],h_N[width][width],h_P[width][width];
	float *d_M,*d_N,*d_P;
	
	printf("\n width= %d \n ", width);
	
	for(i=0;i<width;i++)
	{
		for(j=0;j<width;j++)
			{ 
				h_M[i][j]=1;
				h_N[i][j]=1;
		   }
	}

	

	printf("\nh_M array is: \n");
	for(i=0;i<width;i++)
	{   
		printf("\n");
		for(j=0;j<width;j++)
			printf("%d ",h_M[i][j]);
	}
	
	printf("\n\nh_N array is: \n");
	for(i=0;i<width;i++)
	{   
		printf("\n");
		for(j=0;j<width;j++)
			printf("%d ",h_N[i][j]);
	}
	
	size=sizeof(int)*width*width;
	cudaMalloc((void**)&d_M,size);
	cudaMalloc((void**)&d_N,size);
	cudaMalloc((void**)&d_P,size);
	cudaMemcpy(d_M,h_M,size,cudaMemcpyHostToDevice);
	cudaMemcpy(d_N,h_N,size,cudaMemcpyHostToDevice);

	dim3 dimGrid((width/TILE_WIDTH),(width/TILE_WIDTH),1);
	dim3 dimBlock(TILE_WIDTH,TILE_WIDTH,1);

	MatrixMulKernel<<<dimGrid,dimBlock>>>(d_M,d_N,d_P,width);
	
	cudaMemcpy(h_P,d_P,size,cudaMemcpyDeviceToHost);
	cudaFree(d_M);
	cudaFree(d_N);
	cudaFree(d_P);

	printf("\n\nMultiplied array is: \n");
	for(i=0;i<width;i++)
	{   
		printf("\n");
		for(j=0;j<width;j++)
			printf("%d ",h_P[i][j]);
	}
	
  getch();
  return 0;
}

*/