
#include <hip/hip_runtime.h>
/*
#ifndef __CUDACC__
#define __CUDACC__
#endif
#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include <cuda.h>
#include <stdio.h>
#include <device_functions.h>
#include <string.h>


__global__ void whatever(char  *d_a,char *d_res,int len)
{
	int i=0;
	int flag=1;
	int in=blockIdx.x*blockDim.x+threadIdx.x;


	if(d_a[in]==' ')
		d_res[in]='f';
	else if((d_a[in]>=65 && d_a[in]<=90)||(d_a[in]>=97 && d_a[in]<=122))
		d_res[in]='f';
	else
	{
		for(i=in-1;i>=0 && d_a[i]!=' ';i--)
		{
			if((d_a[i]>=65 && d_a[i]<=90)||(d_a[i]>=97 && d_a[i]<=122))
			{
				flag=0;
				break;
			}
		}

		if(flag==0)
			d_res[in]='f';
		else
			d_res[in]='t';

	}

}



int main()
{
	int i;
	char h_a[1000];
	char h_res[1000];
	char *d_a;
	char *d_res;

	printf("Enter the string: \n");
	gets(h_a);

	//puts(h_a);


	int len=strlen(h_a);

	int size=sizeof(char)*len;

	cudaMalloc((void **)&d_a,size);
	cudaMalloc((void **)&d_res,size);

	cudaMemcpy(d_a,h_a,size,cudaMemcpyHostToDevice);

	whatever<<<1,len>>>(d_a,d_res,len);

	cudaMemcpy(h_res,d_res,size,cudaMemcpyDeviceToHost);

	printf("OUTPUT :\n");

	for(i=0;i<len;i++)
		printf("%c, ",h_res[i]);

	printf("\n\n");

	cudaFree(d_a);
	cudaFree(d_res);


}
*/